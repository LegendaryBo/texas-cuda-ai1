#include "hip/hip_runtime.h"
#include "../struktury/gracz.h"
#include "../struktury/gra.h"
#include "../funkcje/rozgrywka.h"

#include "../funkcje/rezultaty.h"
#include "../funkcje/osobnik.h"
#include "../funkcje/cuda_zlecenia.h"

#include "../struktury/ile_grac_r1.h"
#include "../struktury/ile_grac_rx.h"
#include "../struktury/dobijanie_r1.h"
#include "../struktury/dobijanie_rx.h"
#include "../struktury/stawka_r1.h"
#include "../struktury/stawka_rx.h"
#include "../struktury/czy_grac_r1.h"
#include "../struktury/czy_grac_rx.h"
#include "../struktury/reguly.h"
#include "../struktury/zlecenie.h"


#include "../funkcje/reguly_ilegrac.h"
#include "../funkcje/reguly_stawka.h"
#include "../funkcje/reguly_dobijania.h"
#include "../funkcje/reguly_czygrac.h"
#include "../funkcje/reguly.h"

#include <stdio.h>
#include <stdlib.h>

int rozmiar_genomu = 1234;
//float minimal_bid = 10.0;



void getBilans(Gra *gra,  float *output ) {
	for (int i=0; i < 6; i++)
		output[i] = gra->gracze[i].bilans;
};


Gra *getGraPTR() {
	Gra *gra = new Gra;
	return gra;
};



int getJakisHashcode(int *osobnik, int dimension) {

	int wynik=13;
	for (int i=0; i < dimension; i++) {
		if (getBitHOST(osobnik, i)  == 1)
			wynik = wynik*183 +191;
		else
			wynik = wynik*521 + 31;
	}
	return wynik;

}


Zlecenie **stworzZlecenia(int ktory_nasz, int *osobniki, int N) {

	int licznik=0;

	Zlecenie **zlecenia = (Zlecenie**) malloc(sizeof(Zlecenie*) * N);
	for (int i=0; i < N; i++) {
		zlecenia[i] = noweZlecenie( ktory_nasz,
								  (licznik+1) %100,
								  (licznik+2) %100,
								  (licznik+3) %100,
								  (licznik+4) %100,
								  (licznik+5) %100,
								ktory_nasz, i, osobniki);
		licznik+=6;
	}


	return zlecenia;

}







int **getIndividualPTRPTR(int size) {

	int **wskaznik = (int**)malloc( sizeof(int*) * size );


	return wskaznik;
}

void setIndividualPTR(int *osobnik, int **partnerzy, int index) {
	partnerzy[index] = osobnik;
}


float obliczZlecenia(Zlecenie **zlecenie, int ile_zlecen, int ile_intow) {

	float suma=0.0;

	for (int i=0; i < ile_zlecen; i++) {

		Reguly *regula = getReguly();
		Gra *gra = new Gra;

		nowaGra(   &zlecenie[i]->osobniki[0] + zlecenie[i]->indexOsobnika[0] *  ile_intow ,
				&zlecenie[i]->osobniki[0] + zlecenie[i]->indexOsobnika[1] *  ile_intow,
				&zlecenie[i]->osobniki[0] + zlecenie[i]->indexOsobnika[2] *  ile_intow,
				&zlecenie[i]->osobniki[0] + zlecenie[i]->indexOsobnika[3] *  ile_intow,
				&zlecenie[i]->osobniki[0] + zlecenie[i]->indexOsobnika[4] *  ile_intow ,
				&zlecenie[i]->osobniki[0] + zlecenie[i]->indexOsobnika[5] *  ile_intow ,
				zlecenie[i]->nrRozdania, 3, gra);

		zlecenie[i]->wynik =rozegrajPartieHOST(gra, 0, regula);
		suma+=zlecenie[i]->wynik;
//		printf("gra numer %d - ", i);
//		printf(" %f \n", suma/ile_zlecen);

		free(gra);
	}
	return suma/ile_zlecen;
}


void rozegrajNGier(int ktory_nasz, int **osobniki, float *wynik, int N, int liczba_intow) {

	wynik[0]=0.0;


	int *osobniki_statyczna_tablica = (int*) malloc(sizeof(int) * 101 * liczba_intow);

	for (int i=0; i < 101; i++) {
		for (int j=0; j < liczba_intow; j++)
			osobniki_statyczna_tablica[ j + i * liczba_intow  ] = *(&(osobniki[i])[j]);
	}

	Zlecenie **zlecenia = stworzZlecenia(ktory_nasz, osobniki_statyczna_tablica, N);

	wynik[0] = obliczZlecenia(zlecenia, N, liczba_intow);



	for (int i=0; i < N; i++) {
		free(zlecenia[i]);
	}
	free(zlecenia);

}


//64kb pamieci stalej
static __constant__ int osobniki_const[256*64];

__global__ void obliczZlecenie(int liczbaGier, Zlecenie *zlecenia_cuda, float *wyniki_device, int ile_intow, Gra *gra, Reguly *reguly) {

	//extern __shared__ int temp_count[];

	register int nr_zlecenia;
	nr_zlecenia = blockIdx.x*blockDim.x + threadIdx.x;



	if (nr_zlecenia>=liczbaGier)
	  return;

	(zlecenia_cuda + nr_zlecenia ) -> osobniki = &osobniki_const[0];
	(zlecenia_cuda + nr_zlecenia ) -> nrRozdania = nr_zlecenia;

	(zlecenia_cuda + nr_zlecenia ) -> indexOsobnika[0] = 100;
	(zlecenia_cuda + nr_zlecenia ) -> indexOsobnika[1] = (nr_zlecenia*6 + 1)%100;
	(zlecenia_cuda + nr_zlecenia ) -> indexOsobnika[2] = (nr_zlecenia*6 + 2)%100;
	(zlecenia_cuda + nr_zlecenia ) -> indexOsobnika[3] = (nr_zlecenia*6 + 3)%100;
	(zlecenia_cuda + nr_zlecenia ) -> indexOsobnika[4] = (nr_zlecenia*6 + 4)%100;
	(zlecenia_cuda + nr_zlecenia ) -> indexOsobnika[5] = (nr_zlecenia*6 + 5)%100;

	(zlecenia_cuda + nr_zlecenia ) -> indexGracza = 100;
//	printf("indeks pierwszego osobnika %d \n",(nr_zlecenia*6 + 1)%100);

	//__shared__ Gra gra[4];

	nowaGra(   &osobniki_const[0] + 100 *  ile_intow ,
			&osobniki_const[0] + (nr_zlecenia*6 + 1)%100 *  ile_intow,
			&osobniki_const[0] + (nr_zlecenia*6 + 2)%100 *  ile_intow,
			&osobniki_const[0] + (nr_zlecenia*6 + 3)%100 *  ile_intow,
			&osobniki_const[0] + (nr_zlecenia*6 + 4)%100 *  ile_intow,
			&osobniki_const[0] + (nr_zlecenia*6 + 5)%100 *  ile_intow,
			nr_zlecenia, 3, &gra[nr_zlecenia]);

	float bla = rozegrajPartieDEVICE(&gra[nr_zlecenia], 0, reguly);

//	int spasowani[6];
//	int wygrani[6];
//	for (int i=0; i < 6; i++)
//		spasowani[i]=0;
//	int ile = wygrany(& ( gra[nr_zlecenia].rozdanie ),
//			&spasowani[0],
//			&wygrani[0]);
//
//	float bla=rozegrajPartie(&gra[nr_zlecenia], 0, reguly);
//
//	for  (int i=0; i < ile; i++) {
//		bla += wygrani[i] * wygrani[i];
//	}


//	printf("wynik to %f \n", bla);
	//wyniki_device[ nr_zlecenia ] = bla;

	wyniki_device[ nr_zlecenia ] = bla;


}

// metoda sprawdza, czy wystapil w GPU jakis blad
void obsluzBlad(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) // sprawdzamy, czy blad wystapil
    {
        fprintf(stderr, "Blad Cuda : %s: %s.\n", msg,
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}




void rozegrajNGierCUDA(int ktory_nasz, int **osobniki, float *wynik, int N,
int liczba_intow, int block_size) {


	putenv("CUDA_PROFILE=1");
	putenv("CUDA_PROFILE_LOG=/home/kacper/cudaProfiler/profiler_log");

	putenv("CUDA_PROFILE_CONFIG=/home/kacper/cudaProfiler/cuda_profiler.cfg");

	int *osobniki_statyczna_tablica = (int*) malloc(sizeof(int) * 101 * liczba_intow);

	for (int i=0; i < 101; i++) {
		for (int j=0; j < liczba_intow; j++)
			osobniki_statyczna_tablica[ j + i * liczba_intow  ] = *(&(osobniki[i])[j]);
	}

	//printf("liczba watkow na grid %d\n", block_size);
	//printf("liczba partii %d\n", N);
	//printf("wielkosc osobnika %d\n", liczba_intow*4);

	Gra *gry_cuda;
	size_t size_gry = sizeof(Gra)*N;
	hipMalloc((void **) &gry_cuda, size_gry);

	Reguly *reguly_cuda;
	Reguly *reguly_host = getReguly();
	size_t size_reguly = sizeof(Reguly);
	hipMalloc((void **) &reguly_cuda, size_reguly);



	float *wyniki_device;
	size_t size_wyniki = sizeof(float)*N;
	hipMalloc((void **) &wyniki_device, size_wyniki);
	float *wyniki_host;
	wyniki_host = (float *)malloc( size_wyniki );

	Zlecenie *zlecenia_cuda;
	size_t size_zlecenie = sizeof(Zlecenie)*N;
	hipMalloc((void **) &zlecenia_cuda, size_zlecenie);

    	//int sharedMemSize = block_size * sizeof(int) * 10;

        hipMemcpyToSymbol(HIP_SYMBOL(osobniki_const), osobniki_statyczna_tablica, liczba_intow*sizeof(int)*101);
	obsluzBlad("kopiowanie osobnikow na karte");
	//hipMemcpy(osobniki_cuda, osobniki_statyczna_tablica, liczba_intow*sizeof(int)*101, hipMemcpyHostToDevice);
	hipMemcpy(reguly_cuda, reguly_host, sizeof(Reguly), hipMemcpyHostToDevice);

	int nBlocks = N/block_size + 1;
	if (N%block_size==0)
	  nBlocks--;

	obsluzBlad("kopiowanie pozostalych danych na karte");
	//obliczZlecenie <<< nBlocks, block_size,  sharedMemSize>>> (N, zlecenia_cuda, wyniki_device, liczba_intow, gry_cuda, reguly_cuda);
	hipDeviceSynchronize();
	obsluzBlad("uruchomienia kernela");

	hipMemcpy(wyniki_host, wyniki_device, size_wyniki  , hipMemcpyDeviceToHost);

        obsluzBlad("kopiowanie wynikow z karty");

	float suma = 0.0;
	for (int i=0; i < N; i++) {
		suma += wyniki_host[i];
		//printf("\nCUDA bilans po grze nr %d",sizeof() );
		//printf("\n wskaznik: %d", wyniki_host[i]);
		printf("ma wynik %f ", (suma/N) );
	}

	wynik[0] = suma/N;
}








void destruktorGra(Gra *gra) {
	free(gra);
};

void destruktorInt(int *ptr) {
	free(ptr);
};

void destruktorKodGraya(KodGraya *kodGraya) {
	free(kodGraya);
};

extern void destruktorRozdanie(Rozdanie *rozdanie) {
	free(rozdanie);
};

extern void destruktorHand(Hand *hand) {
	free(hand);
};
